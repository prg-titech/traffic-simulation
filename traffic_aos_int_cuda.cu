#include "hip/hip_runtime.h"
#include <chrono>

#include "traffic_aos_int_cuda.h"
#include "random.h"

using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

namespace simulation {
namespace aos_int {
extern Cell* s_Cell;
extern IndexType s_size_Cell;

extern IndexType* s_outgoing_cells;
extern IndexType s_size_outgoing_cells;

extern IndexType* s_incoming_cells;
extern IndexType s_size_incoming_cells;

extern Car* s_Car;
extern IndexType s_size_Car;

extern IndexType* s_car_paths;
extern IndexType s_size_car_paths;

extern IndexType* s_inactive_cars;
extern IndexType s_size_inactive_cars;

extern TrafficLight* s_TrafficLight;
extern IndexType s_size_TrafficLight;

extern PriorityYieldTrafficController* s_PriorityYieldTrafficController;
extern IndexType s_size_PriorityYieldTrafficController;

extern SharedSignalGroup* s_SharedSignalGroup;
extern IndexType s_size_SharedSignalGroup;

extern IndexType* s_traffic_light_signal_groups;
extern IndexType s_size_traffic_light_signal_groups;

extern IndexType* s_priority_ctrl_signal_groups;
extern IndexType s_size_priority_ctrl_signal_groups;

extern IndexType* s_signal_group_cells;
extern IndexType s_size_signal_group_cells;

extern Simulation* instance;
}  // namespace aos_int

namespace aos_int_cuda {

// Singleton simulation instance.
__device__ Simulation* instance;

// Data storage.
__device__ Cell* s_Cell;
__device__ IndexType s_size_Cell = 0;

__device__ IndexType* s_outgoing_cells;
__device__ IndexType s_size_outgoing_cells = 0;

__device__ IndexType* s_incoming_cells;
__device__ IndexType s_size_incoming_cells = 0;

__device__ Car* s_Car;
__device__ IndexType s_size_Car = 0;

__device__ IndexType* s_car_paths;
__device__ IndexType s_size_car_paths = 0;

__device__ IndexType* s_inactive_cars;
__device__ IndexType s_size_inactive_cars = 0;

__device__ TrafficLight* s_TrafficLight;
__device__ IndexType s_size_TrafficLight = 0;

__device__ PriorityYieldTrafficController* s_PriorityYieldTrafficController;
__device__ IndexType s_size_PriorityYieldTrafficController = 0;

__device__ SharedSignalGroup* s_SharedSignalGroup;
__device__ IndexType s_size_SharedSignalGroup = 0;

__device__ IndexType* s_traffic_light_signal_groups;
__device__ IndexType s_size_traffic_light_signal_groups = 0;

__device__ IndexType* s_priority_ctrl_signal_groups;
__device__ IndexType s_size_priority_ctrl_signal_groups = 0;

__device__ IndexType* s_signal_group_cells;
__device__ IndexType s_size_signal_group_cells = 0;


#define MEMCPY_TO_DEVICE(class, var) \
  class* dev_ ## var; \
  gpuErrchk(hipMalloc((void**) &dev_ ## var, \
             sizeof(class)*simulation::aos_int::s_size_ ## var)); \
  printf("GPU allocation for " STRINGIFY(var) ": %i bytes, %i objects\n", \
         sizeof(class)*simulation::aos_int::s_size_ ## var, \
         simulation::aos_int::s_size_ ## var); \
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(s_ ## var), \
                     &dev_ ## var, sizeof(char*))); \
  gpuErrchk(hipMemcpy((void*) dev_ ## var, \
             (void*) simulation::aos_int::s_ ## var, \
             sizeof(class)*simulation::aos_int::s_size_ ## var, \
             hipMemcpyHostToDevice)); \
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(s_size_ ## var), \
                     &simulation::aos_int::s_size_ ## var, \
                     sizeof(IndexType)));
#define STRINGIFY2(X) #X
#define STRINGIFY(X) STRINGIFY2(X)

void initialize() {
  MEMCPY_TO_DEVICE(Cell, Cell);
  MEMCPY_TO_DEVICE(IndexType, outgoing_cells);
  MEMCPY_TO_DEVICE(IndexType, incoming_cells);
  MEMCPY_TO_DEVICE(Car, Car);
  MEMCPY_TO_DEVICE(IndexType, car_paths);
  MEMCPY_TO_DEVICE(TrafficLight, TrafficLight);
  MEMCPY_TO_DEVICE(PriorityYieldTrafficController,
                   PriorityYieldTrafficController);
  MEMCPY_TO_DEVICE(SharedSignalGroup, SharedSignalGroup);
  MEMCPY_TO_DEVICE(IndexType, traffic_light_signal_groups);
  MEMCPY_TO_DEVICE(IndexType, priority_ctrl_signal_groups);
  MEMCPY_TO_DEVICE(IndexType, signal_group_cells);

  for (int i = 0; i < simulation::aos_int::s_size_Car; ++i) {
    // Fix path_: Contains a pointer.
    auto& car_path = ((Car*) simulation::aos_int::s_Car)[i].path();
    auto path_offset = car_path.buffer()
        - simulation::aos_int::s_car_paths;
    fixed_size_queue<IndexType, false> new_path(dev_car_paths + path_offset,
                                                car_path.capacity());

    gpuErrchk(hipMemcpy((void*) &dev_Car[i].path(), (void*) &new_path,
               sizeof(fixed_size_queue<IndexType, false>),
               hipMemcpyHostToDevice));
  }

  simulation::aos_int::s_size_inactive_cars = simulation::aos_int::s_size_Car;
  MEMCPY_TO_DEVICE(IndexType, inactive_cars);
  simulation::aos_int::s_size_inactive_cars = 0;
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(s_size_inactive_cars),
                     &simulation::aos_int::s_size_inactive_cars,
                     sizeof(IndexType)));

  Simulation* dev_simulation;
  gpuErrchk(hipMalloc((void**) &dev_simulation, sizeof(Simulation)));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(instance), &dev_simulation,
                     sizeof(Simulation*)));
  gpuErrchk(hipMemcpy((void*) dev_simulation,
                       (void*) simulation::aos_int::instance,
                       sizeof(Simulation), hipMemcpyHostToDevice));
}

#undef MEMCPY_TO_DEVICE

__global__ void step_velocity() {
  IndexType id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < s_size_Car) {
    if (s_Car[id].is_active()) {
      s_Car[id].step_velocity();
    }
  }
}

__global__ void step_assert_check_velocity() {
  IndexType id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < s_size_Car) {
    if (s_Car[id].is_active()) {
      s_Car[id].assert_check_velocity();
    }
  }
}

__global__ void step_move() {
  IndexType id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < s_size_Car) {
    if (s_Car[id].is_active()) {
      s_Car[id].step_move();
    }
  }
}

__global__ void step_reactivate() {
  IndexType id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < s_size_Car) {
    s_Car[id].step_reactivate();
  }
}

__global__ void step_traffic_lights() {
  IndexType id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < s_size_TrafficLight) {
    s_TrafficLight[id].step();
  }
}

__global__ void step_priority_ctrl() {
  IndexType id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < s_size_PriorityYieldTrafficController) {
    s_PriorityYieldTrafficController[id].step();
  }
}

__device__ uint64_t s_checksum;

__global__ void checksum_kernel() {
  s_checksum = instance->checksum();
}

__global__ void step_random_state() {
  instance->step_random_state();
}

uint64_t checksum() {
  checksum_kernel<<<1, 1>>>();
  uint64_t result;
  hipMemcpyFromSymbol(&result, HIP_SYMBOL(s_checksum), sizeof(uint64_t));
  return result;
}

void step() {
  IndexType num_cars = simulation::aos_int::s_size_Car;
  IndexType num_traffic_lights = simulation::aos_int::s_size_TrafficLight;
  IndexType num_priority_ctrl =
      simulation::aos_int::s_size_PriorityYieldTrafficController;

  auto t1 = std::chrono::steady_clock::now();

  for (int i = 0; i < 1000; ++i) {
    step_random_state<<<1, 1>>>();
    step_traffic_lights<<<num_traffic_lights / 1024 + 1, 1024>>>();
    step_priority_ctrl<<<num_priority_ctrl / 1024 + 1, 1024>>>();
    step_velocity<<<num_cars / 1024 + 1, 1024>>>();

#ifndef NDEBUG
    step_assert_check_velocity<<<num_cars / 1024 + 1, 1024>>>();
    gpuErrchk(hipDeviceSynchronize());
#endif

    step_move<<<num_cars / 1024 + 1, 1024>>>();
    step_reactivate<<<num_cars / 1024 + 1, 1024>>>();

#ifndef NDEBUG
    gpuErrchk(hipDeviceSynchronize());
#else
    hipDeviceSynchronize();
#endif  // NDEBUG
  }

  auto t2 = std::chrono::steady_clock::now();
  unsigned long millis = std::chrono::duration_cast<std::chrono::milliseconds>(
      t2 - t1).count();
  auto cs = checksum();

  printf("Checksum: %lu, GPU Time (millis): %lu\n", cs, millis);
}

__device__ void Simulation::add_inactive_car(IndexType car) {
  s_inactive_cars[s_size_inactive_cars++] = car;
}

__device__ void Simulation::step_traffic_controllers() {
  assert(false);
}

__device__ IndexType Cell::num_outgoing_cells() const {
  return num_outgoing_cells_;
}

__device__ IndexType Cell::outgoing_cell(IndexType index) const {
  return s_outgoing_cells[first_outgoing_cell_idx_ + index];
}

__device__ IndexType Cell::num_incoming_cells() const {
  return num_incoming_cells_;
}

__device__ IndexType Cell::incoming_cell(IndexType index) const {
  return s_incoming_cells[first_incoming_cell_idx_ + index];
}

__device__ IndexType SharedSignalGroup::num_cells() const {
  return num_cells_;
}

__device__ IndexType SharedSignalGroup::cell(IndexType index) const {
  return s_signal_group_cells[first_cell_idx_ + index];
}

__device__ IndexType TrafficLight::num_signal_groups() const {
  return num_signal_groups_;
}

__device__ IndexType TrafficLight::signal_group(IndexType index) const {
  return s_traffic_light_signal_groups[first_signal_group_idx_ + index];
}

__device__ IndexType PriorityYieldTrafficController::num_groups() const {
  return num_groups_;
}

__device__ IndexType PriorityYieldTrafficController::group(IndexType index)
    const {
  return s_priority_ctrl_signal_groups[first_group_idx_ + index];
}

// Accessor methods for cars.
__device__ IndexType Simulation::num_cars() const { return s_size_Car; }
__device__ IndexType Simulation::car(IndexType index) const { return index; }

// Accessor methods for cells.
__device__ IndexType Simulation::num_cells() const { return s_size_Cell; }
__device__ IndexType Simulation::cell(IndexType index) const { return index; }

// Logic for traffic flow simulation.
#include "traffic_logic.inc"
#include "option_undo.inc"

}  // namespace aos_int
}  // namespace simulation

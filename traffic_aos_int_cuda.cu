#include "hip/hip_runtime.h"
#include <chrono>

#include "traffic_aos_int_cuda.h"
#include "random.h"

#include "cub-1.8.0/cub/util_allocator.cuh"
#include "cub-1.8.0/cub/device/device_radix_sort.cuh"

using namespace hipcub;

using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

namespace simulation {
namespace aos_int {
extern Cell* s_Cell;
extern IndexType s_size_Cell;

extern IndexType* s_outgoing_cells;
extern IndexType s_size_outgoing_cells;

extern IndexType* s_incoming_cells;
extern IndexType s_size_incoming_cells;

extern Car* s_Car;
extern IndexType s_size_Car;

extern IndexType* s_car_paths;
extern IndexType s_size_car_paths;

extern IndexType* s_inactive_cars;
extern IndexType s_size_inactive_cars;

extern TrafficLight* s_TrafficLight;
extern IndexType s_size_TrafficLight;

extern PriorityYieldTrafficController* s_PriorityYieldTrafficController;
extern IndexType s_size_PriorityYieldTrafficController;

extern SharedSignalGroup* s_SharedSignalGroup;
extern IndexType s_size_SharedSignalGroup;

extern IndexType* s_traffic_light_signal_groups;
extern IndexType s_size_traffic_light_signal_groups;

extern IndexType* s_priority_ctrl_signal_groups;
extern IndexType s_size_priority_ctrl_signal_groups;

extern IndexType* s_signal_group_cells;
extern IndexType s_size_signal_group_cells;

extern Simulation* instance;
}  // namespace aos_int

namespace aos_int_cuda {

// Singleton simulation instance.
__device__ Simulation* instance;

// Data storage.
__device__ Cell* s_Cell;
__device__ IndexType s_size_Cell = 0;

__device__ IndexType* s_outgoing_cells;
__device__ IndexType s_size_outgoing_cells = 0;

__device__ IndexType* s_incoming_cells;
__device__ IndexType s_size_incoming_cells = 0;

__device__ Car* s_Car;
__device__ Car* s_tmp_Car;  // temp array used for changing physical order
__device__ IndexType s_size_Car = 0;

__device__ IndexType* s_car_paths;
__device__ IndexType s_size_car_paths = 0;

__device__ IndexType* s_inactive_cars;
__device__ IndexType s_size_inactive_cars = 0;

__device__ TrafficLight* s_TrafficLight;
__device__ IndexType s_size_TrafficLight = 0;

__device__ PriorityYieldTrafficController* s_PriorityYieldTrafficController;
__device__ IndexType s_size_PriorityYieldTrafficController = 0;

__device__ SharedSignalGroup* s_SharedSignalGroup;
__device__ IndexType s_size_SharedSignalGroup = 0;

__device__ IndexType* s_traffic_light_signal_groups;
__device__ IndexType s_size_traffic_light_signal_groups = 0;

__device__ IndexType* s_priority_ctrl_signal_groups;
__device__ IndexType s_size_priority_ctrl_signal_groups = 0;

__device__ IndexType* s_signal_group_cells;
__device__ IndexType s_size_signal_group_cells = 0;

__device__ IndexType* s_Car_reorder_in;
__device__ IndexType* s_Car_reorder_keys_in;

__device__ IndexType* s_Car_reorder;
__device__ IndexType* s_Car_reorder_keys;

IndexType* d_car_reorder_in;
IndexType* d_car_reorder_keys_in;
IndexType* d_car_reorder;
IndexType* d_car_reorder_keys;

#define MEMCPY_TO_DEVICE(class, var) \
  class* dev_ ## var; \
  gpuErrchk(hipMalloc((void**) &dev_ ## var, \
             sizeof(class)*simulation::aos_int::s_size_ ## var)); \
  printf("GPU allocation for " STRINGIFY(var) ": %i bytes, %i objects\n", \
         sizeof(class)*simulation::aos_int::s_size_ ## var, \
         simulation::aos_int::s_size_ ## var); \
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(s_ ## var), \
                     &dev_ ## var, sizeof(char*))); \
  gpuErrchk(hipMemcpy((void*) dev_ ## var, \
             (void*) simulation::aos_int::s_ ## var, \
             sizeof(class)*simulation::aos_int::s_size_ ## var, \
             hipMemcpyHostToDevice)); \
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(s_size_ ## var), \
                     &simulation::aos_int::s_size_ ## var, \
                     sizeof(IndexType)));
#define STRINGIFY2(X) #X
#define STRINGIFY(X) STRINGIFY2(X)

void initialize() {
  MEMCPY_TO_DEVICE(Cell, Cell);
  MEMCPY_TO_DEVICE(IndexType, outgoing_cells);
  MEMCPY_TO_DEVICE(IndexType, incoming_cells);
  MEMCPY_TO_DEVICE(Car, Car);
  MEMCPY_TO_DEVICE(IndexType, car_paths);
  MEMCPY_TO_DEVICE(TrafficLight, TrafficLight);
  MEMCPY_TO_DEVICE(PriorityYieldTrafficController,
                   PriorityYieldTrafficController);
  MEMCPY_TO_DEVICE(SharedSignalGroup, SharedSignalGroup);
  MEMCPY_TO_DEVICE(IndexType, traffic_light_signal_groups);
  MEMCPY_TO_DEVICE(IndexType, priority_ctrl_signal_groups);
  MEMCPY_TO_DEVICE(IndexType, signal_group_cells);

  for (int i = 0; i < simulation::aos_int::s_size_Car; ++i) {
    // Fix path_: Contains a pointer.
    auto& car_path = ((Car*) simulation::aos_int::s_Car)[i].path();
    auto path_offset = car_path.buffer()
        - simulation::aos_int::s_car_paths;
    fixed_size_queue<IndexType, false> new_path(dev_car_paths + path_offset,
                                                car_path.capacity());

    gpuErrchk(hipMemcpy((void*) &dev_Car[i].path(), (void*) &new_path,
               sizeof(fixed_size_queue<IndexType, false>),
               hipMemcpyHostToDevice));
  }

  simulation::aos_int::s_size_inactive_cars = simulation::aos_int::s_size_Car;
  MEMCPY_TO_DEVICE(IndexType, inactive_cars);
  simulation::aos_int::s_size_inactive_cars = 0;
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(s_size_inactive_cars),
                     &simulation::aos_int::s_size_inactive_cars,
                     sizeof(IndexType)));

  Simulation* dev_simulation;
  gpuErrchk(hipMalloc((void**) &dev_simulation, sizeof(Simulation)));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(instance), &dev_simulation,
                     sizeof(Simulation*)));
  gpuErrchk(hipMemcpy((void*) dev_simulation,
                       (void*) simulation::aos_int::instance,
                       sizeof(Simulation), hipMemcpyHostToDevice));

  IndexType* h_car_reorder = new IndexType[simulation::aos_int::s_size_Car];
  for (int i = 0; i < simulation::aos_int::s_size_Car; ++i) {
    h_car_reorder[i] = i;
  }
  hipMalloc((void**) &d_car_reorder, sizeof(IndexType)*simulation::aos_int::s_size_Car);
  hipMemcpyToSymbol(HIP_SYMBOL(s_Car_reorder), &d_car_reorder, sizeof(IndexType*));
  hipMemcpy(d_car_reorder, h_car_reorder, sizeof(IndexType)*simulation::aos_int::s_size_Car,
             hipMemcpyHostToDevice);
  hipMalloc((void**) &d_car_reorder_keys, sizeof(IndexType)*simulation::aos_int::s_size_Car);
  hipMemcpyToSymbol(HIP_SYMBOL(s_Car_reorder_keys), &d_car_reorder_keys, sizeof(IndexType*));

  hipMalloc((void**) &d_car_reorder_in, sizeof(IndexType)*simulation::aos_int::s_size_Car);
  hipMemcpyToSymbol(HIP_SYMBOL(s_Car_reorder_in), &d_car_reorder_in, sizeof(IndexType*));

  hipMalloc((void**) &d_car_reorder_keys_in, sizeof(IndexType)*simulation::aos_int::s_size_Car);
  hipMemcpyToSymbol(HIP_SYMBOL(s_Car_reorder_keys_in), &d_car_reorder_keys_in, sizeof(IndexType*));

  Car* d_tmp_car;
  hipMalloc((void**) &d_tmp_car, sizeof(Car)*simulation::aos_int::s_size_Car);
  hipMemcpyToSymbol(HIP_SYMBOL(s_tmp_Car), &d_tmp_car, sizeof(Car*));

  gpuErrchk(hipDeviceSynchronize());
}

#undef MEMCPY_TO_DEVICE

#define PHYSICAL_REORDER true

__global__ void step_velocity() {
  IndexType id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < s_size_Car) {
    int realid = PHYSICAL_REORDER ? id : s_Car_reorder[id];
    if (s_Car[realid].is_active()) {
      s_Car[realid].step_velocity();
    }
  }
}

__global__ void step_assert_check_velocity() {
  IndexType id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < s_size_Car) {
    int realid = PHYSICAL_REORDER ? id : s_Car_reorder[id];
    if (s_Car[realid].is_active()) {
      s_Car[realid].assert_check_velocity();
    }
  }
}

__global__ void step_move() {
  IndexType id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < s_size_Car) {
    int realid = PHYSICAL_REORDER ? id : s_Car_reorder[id];
    if (s_Car[realid].is_active()) {
      s_Car[realid].step_move();
    }
  }
}

__global__ void step_reactivate() {
  IndexType id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < s_size_Car) {
    int realid = PHYSICAL_REORDER ? id : s_Car_reorder[id];
    s_Car[realid].step_reactivate();
  }
}

__global__ void step_traffic_lights() {
  IndexType id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < s_size_TrafficLight) {
    s_TrafficLight[id].step();
  }
}

__global__ void step_priority_ctrl() {
  IndexType id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < s_size_PriorityYieldTrafficController) {
    s_PriorityYieldTrafficController[id].step();
  }
}

__global__ void step_prepare_reorder() {
  IndexType id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < s_size_Car) {
    s_Car_reorder_in[id] = id;
    s_Car_reorder_keys_in[id] = s_Car[id].velocity();
//    s_Car_reorder_keys_in[id] = (s_Car[id].velocity() + 1) * s_size_Car + id;
//    s_Car_reorder_keys_in[id] = s_Cell[s_Car[id].position()].type();
//    s_Car_reorder_keys_in[id] = s_Cell[s_Car[id].position()].type()* s_size_Car + id;
//    s_Car_reorder_keys_in[id] = s_Car[id].rand32();

  }
}

__global__ void step_physical_reorder() {
  IndexType id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < s_size_Car) {
    //hipMemcpy(&s_tmp_Car[id], &s_Car[s_Car_reorder[id]], sizeof(Car), hipMemcpyDeviceToDevice);
    //memcpy(&s_tmp_Car[id], &s_Car[s_Car_reorder[id]], sizeof(Car));
    int* target = (int*) (&s_tmp_Car[id]);
    int* from = (int*) (&s_Car[id]);
    for (int i = 0; i < sizeof(Car)/sizeof(int); ++i) {
      *(target + i) = *(from + i);
    }
  }
}

__global__ void step_swap_car_arrays() {
  Car* tmp = s_Car;
  s_Car = s_tmp_Car;
  s_tmp_Car = tmp;
}

__device__ uint64_t s_checksum;

__global__ void checksum_kernel() {
  s_checksum = instance->checksum();
}

__global__ void step_random_state() {
  instance->step_random_state();
}

uint64_t checksum() {
  checksum_kernel<<<1, 1>>>();
  uint64_t result;
  hipMemcpyFromSymbol(&result, HIP_SYMBOL(s_checksum), sizeof(uint64_t));
  return result;
}

#define BLOCK_S 768

void step_reorder() {
  IndexType num_cars = simulation::aos_int::s_size_Car;
  step_prepare_reorder<<<num_cars / BLOCK_S + 1, BLOCK_S>>>();

  // Determine temporary device storage requirements
  void     *d_temp_storage = NULL;
  size_t   temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
      d_car_reorder_keys_in, d_car_reorder_keys, d_car_reorder_in, d_car_reorder, num_cars);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
      d_car_reorder_keys_in, d_car_reorder_keys, d_car_reorder_in, d_car_reorder, num_cars);
  // d_keys_out            <-- [0, 3, 5, 6, 7, 8, 9]
  // d_values_out          <-- [5, 4, 3, 1, 2, 0, 6]


  // Now physical reordering
  if (PHYSICAL_REORDER) {
    step_physical_reorder<<<num_cars / BLOCK_S + 1, BLOCK_S>>>();
    step_swap_car_arrays<<<1, 1>>>();
  }

  gpuErrchk(hipDeviceSynchronize());
}

void step() {
  printf("CAR SIZE: %i\n", sizeof(Car));

  IndexType num_cars = simulation::aos_int::s_size_Car;
  IndexType num_traffic_lights = simulation::aos_int::s_size_TrafficLight;
  IndexType num_priority_ctrl =
      simulation::aos_int::s_size_PriorityYieldTrafficController;

  auto t1 = std::chrono::steady_clock::now();
  unsigned long reorder_time =0;

  for (int i = 0; i < 1000; ++i) {
    step_random_state<<<1, 1>>>();
    step_traffic_lights<<<num_traffic_lights / BLOCK_S + 1, BLOCK_S>>>();
    step_priority_ctrl<<<num_priority_ctrl / BLOCK_S + 1, BLOCK_S>>>();
    step_velocity<<<num_cars / BLOCK_S + 1, BLOCK_S>>>();

#ifndef NDEBUG
    step_assert_check_velocity<<<num_cars / BLOCK_S + 1, BLOCK_S>>>();
    gpuErrchk(hipDeviceSynchronize());
#endif

    step_move<<<num_cars / BLOCK_S + 1, BLOCK_S>>>();
    step_reactivate<<<num_cars / BLOCK_S + 1, BLOCK_S>>>();

    if (false && i % 5 == 0 ) {
      auto t3 = std::chrono::steady_clock::now();
      step_reorder();
      auto t4 = std::chrono::steady_clock::now();

      reorder_time += std::chrono::duration_cast<std::chrono::milliseconds>(
        t4 - t3).count();
    }
#ifndef NDEBUG
    gpuErrchk(hipDeviceSynchronize());
#else
    hipDeviceSynchronize();
#endif  // NDEBUG
  }

  auto t2 = std::chrono::steady_clock::now();
  unsigned long millis = std::chrono::duration_cast<std::chrono::milliseconds>(
      t2 - t1).count();
  auto cs = checksum();

  printf("Checksum: %lu, GPU Time (millis): %lu, Reorder time: %lu\n", cs, millis,reorder_time);
}

__device__ void Simulation::add_inactive_car(IndexType car) {
  s_inactive_cars[s_size_inactive_cars++] = car;
}

__device__ void Simulation::step_traffic_controllers() {
  assert(false);
}

__device__ IndexType Cell::num_outgoing_cells() const {
  return num_outgoing_cells_;
}

__device__ IndexType Cell::outgoing_cell(IndexType index) const {
  return s_outgoing_cells[first_outgoing_cell_idx_ + index];
}

__device__ IndexType Cell::num_incoming_cells() const {
  return num_incoming_cells_;
}

__device__ IndexType Cell::incoming_cell(IndexType index) const {
  return s_incoming_cells[first_incoming_cell_idx_ + index];
}

__device__ IndexType SharedSignalGroup::num_cells() const {
  return num_cells_;
}

__device__ IndexType SharedSignalGroup::cell(IndexType index) const {
  return s_signal_group_cells[first_cell_idx_ + index];
}

__device__ IndexType TrafficLight::num_signal_groups() const {
  return num_signal_groups_;
}

__device__ IndexType TrafficLight::signal_group(IndexType index) const {
  return s_traffic_light_signal_groups[first_signal_group_idx_ + index];
}

__device__ IndexType PriorityYieldTrafficController::num_groups() const {
  return num_groups_;
}

__device__ IndexType PriorityYieldTrafficController::group(IndexType index)
    const {
  return s_priority_ctrl_signal_groups[first_group_idx_ + index];
}

// Accessor methods for cars.
__device__ IndexType Simulation::num_cars() const { return s_size_Car; }
__device__ IndexType Simulation::car(IndexType index) const { return index; }

// Accessor methods for cells.
__device__ IndexType Simulation::num_cells() const { return s_size_Cell; }
__device__ IndexType Simulation::cell(IndexType index) const { return index; }

// Logic for traffic flow simulation.
#include "traffic_logic.inc"
#include "option_undo.inc"

}  // namespace aos_int
}  // namespace simulation

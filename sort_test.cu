/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Simple example of DeviceRadixSort::SortPairs().
 *
 * Sorts an array of float keys paired with a corresponding array of int values.
 *
 * To compile using the command line:
 *   nvcc -arch=sm_XX example_device_radix_sort.cu -I../.. -lcudart -O3
 *
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>
#include <algorithm>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#include "cub-1.8.0/cub/util_allocator.cuh"
#include "cub-1.8.0/cub/device/device_radix_sort.cuh"

using namespace hipcub;
using namespace std;

int main() {
  // Declare, allocate, and initialize device-accessible pointers for sorting data
  int h_arr_keys[] = {1, 3, 6, 1, 8, 4, 19, 0, 0, 1, 45, 2, 4, 2, 2, 9};
  int h_arr_vals[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16};

  int  num_items = 16;          // e.g., 7
  int  *d_keys_in;         // e.g., [8, 6, 7, 5, 3, 0, 9]
  int  *d_keys_out;        // e.g., [        ...        ]
  int  *d_values_in;       // e.g., [0, 1, 2, 3, 4, 5, 6]
  int  *d_values_out;      // e.g., [        ...        ]
  
  hipMalloc((void**) &d_keys_in, 4*16);
  hipMalloc((void**) &d_keys_out, 4*16);
  hipMalloc((void**) &d_values_in, 4*16);
  hipMalloc((void**) &d_values_out, 4*16);
  gpuErrchk(hipDeviceSynchronize());

  hipMemcpy(d_keys_in, h_arr_keys, 4*16, hipMemcpyHostToDevice);
  hipMemcpy(d_values_in, h_arr_vals, 4*16, hipMemcpyHostToDevice);
  gpuErrchk(hipDeviceSynchronize());

  // Determine temporary device storage requirements
  void     *d_temp_storage = NULL;
  size_t   temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
      d_keys_in, d_keys_out, d_values_in, d_values_out, num_items);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
      d_keys_in, d_keys_out, d_values_in, d_values_out, num_items);
  // d_keys_out            <-- [0, 3, 5, 6, 7, 8, 9]
  // d_values_out          <-- [5, 4, 3, 1, 2, 0, 6]
  gpuErrchk(hipDeviceSynchronize());

  hipMemcpy(h_arr_keys, d_keys_out, 4*16, hipMemcpyDeviceToHost);
  hipMemcpy(h_arr_vals, d_values_out, 4*16, hipMemcpyDeviceToHost);
  gpuErrchk(hipDeviceSynchronize());

  for(int i = 0; i < 16; ++i) {
    printf("%i         %i\n", h_arr_keys[i], h_arr_vals[i]);
  }

}
